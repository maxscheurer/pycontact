#include "hip/hip_runtime.h"
#include <sasaKernel.cu>
#include <sasaCuda.hh>
#include <assert.h>
#include <iostream>
#include <chTimer.hpp>


double calculate_sasa_cuda( float *pos, int natoms, float pairdist,  float *radius,  int npts,
                           float srad, int pointstyle, int restricted,  int* restrictedList) {

    const int blockSize = 256;
    const int gridSize = ceil(static_cast<float>(natoms) / static_cast<float>(blockSize));
    const int neighboursInShMem = 3000;

    //std::cout << "Grid Size: " << gridSize << std::endl;
    //std::cout << "Block Size: " << blockSize << std::endl;

    float* h_sasa = NULL;
    float* d_sasa = NULL;
    float3* d_pos = NULL;
    float* d_radius = NULL;
    float3* h_points = NULL;
    float3* d_points = NULL;

    ChTimer kernelTimer;


    h_sasa = static_cast<float*>(malloc(static_cast<size_t>(natoms * sizeof(*h_sasa))));
    h_points = static_cast<float3*>(malloc(static_cast<size_t>(npts * sizeof(*h_points))));
    hipMalloc(&d_sasa, static_cast<size_t>(natoms * sizeof(*d_sasa)));
    hipMalloc(&d_pos, static_cast<size_t>(natoms * sizeof(*d_pos)));
    hipMalloc(&d_radius, static_cast<size_t>(natoms * sizeof(*d_radius)));
    hipMalloc(&d_points, static_cast<size_t>(npts * sizeof(*d_points)));

    memset(h_sasa, 0.0f, natoms * sizeof(*h_sasa));

    float phi_k = 0.0f;
    for (int k = 1; k <= npts; k++) {
        float h_k = 2.0f * (k - 1.0f) / (npts - 1.0f) - 1.0f;
        float theta_k = acosf(h_k);
        if (k == 1 || k == npts) {
            phi_k = 0.0f;
        } else {
            phi_k = fmod((phi_k + 3.6f / sqrtf(npts * (1.0f - h_k*h_k))), (float)(2.0f*M_PI));
        }
        h_points[k-1].x = cosf(phi_k) * sinf(theta_k);
        h_points[k-1].y = sinf(phi_k) * sinf(theta_k);
        h_points[k-1].z = cosf(theta_k);
    }

    if (d_pos == NULL || d_radius == NULL || d_sasa == NULL || d_points == NULL || h_points == NULL || h_sasa == NULL) {
        std::cout << "\033[31m***" << std::endl
            << "*** Error - Allocation of Memory failed!!!" << std::endl
            << "***\033[0m" << std::endl;
    }

    hipMemcpy(d_pos, pos, static_cast<size_t>(natoms * sizeof(*d_pos)), hipMemcpyHostToDevice);
    hipMemcpy(d_radius, radius, static_cast<size_t>(natoms * sizeof(*d_radius)), hipMemcpyHostToDevice);
    hipMemcpy(d_sasa, h_sasa, static_cast<size_t>(natoms * sizeof(*d_sasa)), hipMemcpyHostToDevice);
    hipMemcpy(d_points, h_points, static_cast<size_t>(npts * sizeof(*d_points)), hipMemcpyHostToDevice);

    kernelTimer.start();

    SasaKernel<<< gridSize, blockSize, neighboursInShMem * sizeof(float4) >>>(natoms, pairdist,
        npts, neighboursInShMem, srad, d_pos, d_radius, d_points, d_sasa);
    hipDeviceSynchronize();

    kernelTimer.stop();

    hipError_t hipError_t = hipGetLastError();
    if ( hipError_t != hipSuccess ) {
        std::cout << "\033[31m***" << std::endl
                  << "***ERROR*** " << hipError_t << " - " << hipGetErrorString(hipError_t)
                    << std::endl
                  << "***\033[0m" << std::endl;
    }

    hipMemcpy(h_sasa, d_sasa, static_cast<size_t>(natoms * sizeof(*d_sasa)), hipMemcpyDeviceToHost);
    float sasa = 0.0f;
    for (int i = 0; i < natoms; i++) {
      sasa += h_sasa[i];
    }
    std::cout << "SASA: " <<  sasa << std::endl;
    std::cout << "Kernel time: " << kernelTimer.getTime() << std::endl;
    std::cout << natoms << std::endl;
    free(h_sasa);
    free(h_points);
    hipFree(d_sasa);
    hipFree(d_pos);
    hipFree(d_radius);
    hipFree(d_points);


    return sasa;
}
