
#include <hip/hip_runtime.h>
#include <stdio.h>

void __global__ SasaKernel(int natoms, float pairdist, const int npts,
    const int neighboursInSmem, float srad, float3* pos,
    float* radius, float3* points, float* sasa)
{
    extern __shared__ float4 s_mem[];
    const int thread = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < neighboursInSmem; i++) {
        float4 atom;
        atom.x = pos[i].x;
        atom.y = pos[i].y;
        atom.z = pos[i].z;
        atom.w = radius[i];
        s_mem[i] = atom;
    }
    __syncthreads();

    if (thread < natoms) {
        float3 center = pos[thread];
        float currentRadius = radius[thread];

        float3 scaledPoint;
        int remainingPoints = npts;

        for (int i = 0; i < npts; i++) {
            scaledPoint.x = points[i].x * (currentRadius + srad) + center.x;
            scaledPoint.y = points[i].y * (currentRadius + srad) + center.y;
            scaledPoint.z = points[i].z * (currentRadius + srad) + center.z;

            bool overlap = false;

            for (int atomId = 0; atomId < natoms; atomId++) {
                if (atomId != thread) {
                    if (atomId < neighboursInSmem) {
                        float4 neighbour = s_mem[atomId];
                        float neighbourRad2 = neighbour.w + srad;
                        neighbourRad2 *= neighbourRad2;
                        float3 dr;
                        dr.x = scaledPoint.x - neighbour.x;
                        dr.y = scaledPoint.y - neighbour.y;
                        dr.z = scaledPoint.z - neighbour.z;

                        if (dr.x*dr.x + dr.y*dr.y + dr.z*dr.z <= neighbourRad2) {
                            overlap = true;
                            break;
                        }
                    } else {
                        float neighbourRad2 = radius[atomId] + srad;
                        neighbourRad2 *= neighbourRad2;
                        float3 neighbourCenter = pos[atomId];
                        float3 dr;
                        dr.x = scaledPoint.x - neighbourCenter.x;
                        dr.y = scaledPoint.y - neighbourCenter.y;
                        dr.z = scaledPoint.z - neighbourCenter.z;

                        if (dr.x*dr.x + dr.y*dr.y + dr.z*dr.z <= neighbourRad2) {
                            overlap = true;
                            break;
                        }
                    }
                }
            }
            if (overlap) {
                remainingPoints--;
            }
        }
        sasa[thread] = 12.5663706144 * powf(currentRadius+srad, 2) * (float)(remainingPoints) / npts;
    }
}
